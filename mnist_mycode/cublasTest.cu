
#include <hip/hip_runtime.h>
//#include <iostream>
//#include<cuda.h>
//#include <cudnn.h>
//#include <Windows.h>
//#include <cublas.h>
//#include <cublas_v2.h>
//
//using namespace std;
//
//void checkCUDNN(cudnnStatus_t status)
//{
//	if (status != CUDNN_STATUS_SUCCESS)
//		cout << "[ERROR] CUDNN" << status << endl;
//}
//
//void checkCUDA(cudaError_t error)
//{
//	if (error != CUDA_SUCCESS)
//		cout << "[ERROR] CUDA" << error << endl;
//}
//
//void print(char* title, float* src, int filter_num, int h, int w)
//{
//	cout << title << endl;
//	for (int i = 0; i < filter_num; i++) {
//		for (int y = 0; y < h; y++) {
//			for (int x = 0; x < w; x++) {
//				printf("%.0f ", src[i*h*w + y*w + x]);
//			}
//			cout << endl;
//		}
//		cout << endl;
//	}
//}
//
//int main()
//{
//	const int batch_count = 1; //입력 데이터 개수, 배치사이즈
//	const int in_channel = 2; //입력 데이터 채널 수
//	const int in_height = 4;// 입력 데이터 세로 길이
//	const int in_width = 4;// 입력 데이터 가로 길이
//	const int out_channel = 2; //출력 클래스 수
//	const int filter_width = 3; //컨벌루션 필터 가로 길이
//	const int filter_height = 3; // 컨벌루션 필터 세로 길이
//	const int filter_num = 1; //컨벌루션 필터 개수
//	const int padding_w = 1; //컨벌루션 패딩.
//	const int padding_h = 1;
//	const int stride_horizontal = 1;
//	const int stride_vertical = 1;
//	const int pool_window_w = 2;
//	const int pool_window_h = 2;
//	const int pool_padding_horizontal = 0;
//	const int pool_padding_vertical = 0;
//	const int pool_stride_horizontal = 2;
//	const int pool_stride_vertical = 2;
//	const int pool_w = in_width / pool_stride_horizontal;
//	const int pool_h = in_height / pool_stride_vertical;
//	const int src_len = batch_count*filter_num*in_height*in_width;
//	const int pool_len = batch_count*filter_num*pool_h*pool_w;
//	float inData_NCHW[batch_count][in_channel][in_height][in_width];
//	float inData_NHWC[batch_count][in_height][in_width][in_channel];
//	float outData[batch_count][filter_num][in_height][in_width];
//	float *inData_d;
//	float *outData_d, *outData1_d;
//	float *filterData_d; // device 컨벌루션 필터 데이터
//	float *filterData2_d; // device FCN 필터 데이터
//	float *biasData_d;
//	float *hostArray = new float[src_len];
//	void* workSpace; //cuDNN 작업 중 사용할 버퍼 메모리.
//
//
//
//					 //입력 데이터 셋팅
//	for (int i = 0; i < in_channel; i++) {
//		for (int y = 0; y < in_height; y++) {
//			for (int x = 0; x < in_width; x++) {
//				inData_NCHW[0][i][y][x] = i * in_channel * in_width*in_height + y*in_height + x;
//			}
//		}
//	}
//
//	//입력 데이터 형태 변환
//	for (int i = 0; i < in_channel; i++) {
//		for (int y = 0; y < in_height; y++) {
//			for (int x = 0; x < in_width; x++) {
//				inData_NHWC[0][y][x][i] = inData_NCHW[0][i][y][x];
//			}
//		}
//	}
//
//	//필터 셋팅
//	float filterData[filter_num][in_channel][filter_height][filter_width] = {
//		{ { { 0.0f, 0.0f, 0.0f },{ 0.0f, 1.0f, 0.0f },{ 0.0f, 0.0f, 0.0f } },
//		{ { 0.0f, 0.0f, 0.0f },{ 0.0f, 0.0f, 1.0f },{ 0.0f, 0.0f, 0.0f } }
//		}
//	};
//
//	//FCN 가중치
//	//float filterData2[out_channel][filter_num][pool_h][pool_w] = { { { { 0.1f, 0.1f },{ 0.1f, 0.1f } } } ,{ { { 0.2f, 0.2f },{ 0.2f, 0.2f } } } };
//	
//	float filterData2[out_channel * pool_h * pool_w] = { 0.1f, 0.1f, 0.1f, 0.1f, 0.2f, 0.2f, 0.2f, 0.2f };
//
//	float biasData[filter_num] = { -20 };
//
//	cout << "in_NCHW" << endl;
//	for (int i = 0; i < in_channel; i++) {
//		for (int y = 0; y < in_height; y++) {
//			for (int x = 0; x < in_width; x++) {
//				printf("%.1f ", inData_NCHW[0][i][y][x]);
//			} cout << endl;
//		} cout << endl;
//	}
//
//	cout << "in_NHWC" << endl;
//	for (int y = 0; y < in_height; y++) {
//		for (int x = 0; x < in_width; x++) {
//			for (int i = 0; i < in_channel; i++) {
//				printf("%.1f ", inData_NHWC[0][y][x][i]);
//			} cout << endl;
//		} cout << endl;
//	}
//
//	cout << "weights" << endl;
//	for (int i = 0; i < in_channel; i++) {
//		for (int y = 0; y < filter_height; y++) {
//			for (int x = 0; x < filter_width; x++) {
//				printf("%.1f ", filterData[0][i][y][x]);
//			} cout << endl;
//		}cout << endl;
//	}
//
//	//GPU 메모리 할당
//	checkCUDA(cudaMalloc((void**)&inData_d, sizeof(inData_NHWC)));
//	checkCUDA(cudaMalloc((void**)&outData_d, sizeof(outData)));
//	checkCUDA(cudaMalloc((void**)&filterData_d, sizeof(filterData)));
//	checkCUDA(cudaMalloc((void**)&filterData2_d, sizeof(filterData2)));
//	checkCUDA(cudaMalloc((void**)&outData1_d, sizeof(outData)));
//	checkCUDA(cudaMalloc((void**)&biasData_d, sizeof(biasData)));
//
//	//CPU에서 GPU로 복사
//	checkCUDA(cudaMemcpy(inData_d, inData_NHWC, sizeof(inData_NHWC), cudaMemcpyHostToDevice));
//	checkCUDA(cudaMemcpy(filterData_d, filterData, sizeof(filterData), cudaMemcpyHostToDevice));
//	checkCUDA(cudaMemcpy(filterData2_d, filterData2, sizeof(filterData2), cudaMemcpyHostToDevice));
//	checkCUDA(cudaMemcpy(biasData_d, biasData, sizeof(biasData), cudaMemcpyHostToDevice));
//
//	
//
//	//CUDNN 배열
//	cudnnHandle_t cudnnHandle; // cuDNN 핸들러
//	cudnnTensorDescriptor_t inTensorDesc, outTensorDesc, biasTensorDesc, poolOutTensorDesc, sftTensorDesc; //데이터 구조체 선언
//	cudnnFilterDescriptor_t filterDesc, filterDesc2;
//	cudnnConvolutionDescriptor_t convDesc, convDesc2;
//	cudnnPoolingDescriptor_t poolDesc;
//	cudnnActivationDescriptor_t actDesc; //활성함수 구조체 선언
//
//	cublasHandle_t cublasHandle;
//
//										 //할당
//	checkCUDNN(cudnnCreate(&cudnnHandle));
//	checkCUDNN(cudnnCreateTensorDescriptor(&inTensorDesc));
//	checkCUDNN(cudnnCreateTensorDescriptor(&outTensorDesc));
//	checkCUDNN(cudnnCreateTensorDescriptor(&biasTensorDesc));
//	checkCUDNN(cudnnCreateTensorDescriptor(&poolOutTensorDesc));
//	checkCUDNN(cudnnCreateTensorDescriptor(&sftTensorDesc));
//	checkCUDNN(cudnnCreateFilterDescriptor(&filterDesc));
//	checkCUDNN(cudnnCreateFilterDescriptor(&filterDesc2));
//	checkCUDNN(cudnnCreateConvolutionDescriptor(&convDesc));
//	checkCUDNN(cudnnCreateConvolutionDescriptor(&convDesc2));
//	checkCUDNN(cudnnCreatePoolingDescriptor(&poolDesc));
//	checkCUDNN(cudnnCreateActivationDescriptor(&actDesc));
//
//	cublasCreate(&cublasHandle);
//
//	cublasSetMatrix(out_channel, pool_h * pool_w, sizeof(*filterData2), filterData2, 2, filterData2_d, 2);
//
//	//초기화
//	//inData_NHWC정보
//	checkCUDNN(cudnnSetTensor4dDescriptor(inTensorDesc, CUDNN_TENSOR_NHWC, CUDNN_DATA_FLOAT, batch_count, in_channel, in_height, in_width));
//	checkCUDNN(cudnnSetFilter4dDescriptor(filterDesc, CUDNN_DATA_FLOAT, CUDNN_TENSOR_NCHW, filter_num, in_channel, filter_height, filter_width));
//	checkCUDNN(cudnnSetFilter4dDescriptor(filterDesc2, CUDNN_DATA_FLOAT, CUDNN_TENSOR_NCHW, out_channel, filter_num, pool_h, pool_w));
//	checkCUDNN(cudnnSetConvolution2dDescriptor(convDesc, padding_h, padding_w, stride_vertical, stride_horizontal, 1, 1, CUDNN_CROSS_CORRELATION, CUDNN_DATA_FLOAT));
//	//FCN 정보
//	checkCUDNN(cudnnSetConvolution2dDescriptor(convDesc2, 0, 0, 2, 2, 1, 1, CUDNN_CROSS_CORRELATION, CUDNN_DATA_FLOAT));
//	checkCUDNN(cudnnSetPooling2dDescriptor(poolDesc, CUDNN_POOLING_MAX, CUDNN_PROPAGATE_NAN, pool_window_h, pool_window_w, pool_padding_vertical, pool_padding_horizontal, pool_stride_vertical
//		, pool_stride_horizontal));
//	checkCUDNN(cudnnSetTensor4dDescriptor(biasTensorDesc, CUDNN_TENSOR_NCHW, CUDNN_DATA_FLOAT, 1, filter_num, 1, 1));
//	checkCUDNN(cudnnSetActivationDescriptor(actDesc, CUDNN_ACTIVATION_RELU, CUDNN_PROPAGATE_NAN, 0));
//
//	int out_n, out_c, out_h, out_w;
//	//입력데이터를 위 셋팅대로 컨벌루션을 했을 때 출력 구조
//	checkCUDNN(cudnnGetConvolution2dForwardOutputDim(convDesc, inTensorDesc, filterDesc, &out_n, &out_c, &out_h, &out_w));
//	printf("conv out shape (n x c x h x w) = (%d x %d x %d x %d)\n", out_n, out_c, out_h, out_w);
//	checkCUDNN(cudnnSetTensor4dDescriptor(outTensorDesc, CUDNN_TENSOR_NCHW, CUDNN_DATA_FLOAT, out_n, out_c, out_h, out_w));
//
//	//풀링 결과 구조 확인
//	checkCUDNN(cudnnGetPooling2dForwardOutputDim(poolDesc, outTensorDesc, &out_n, &out_c, &out_h, &out_w));
//	printf("pool out shape (n x c x h x w) = (%d x %d x %d x %d)\n", out_n, out_c, out_h, out_w);
//	//풀링 결과 구조 셋업
//	checkCUDNN(cudnnSetTensor4dDescriptor(poolOutTensorDesc, CUDNN_TENSOR_NCHW, CUDNN_DATA_FLOAT, out_n, out_c, out_h, out_w));
//
//	//FCN 결과 구조 확인
//	checkCUDNN(cudnnGetConvolution2dForwardOutputDim(convDesc2, poolOutTensorDesc, filterDesc2, &out_n, &out_c, &out_h, &out_w));
//	printf("FCN out shape (n x c x h x w)= (%d x %d x %d x %d)\n", out_n, out_c, out_h, out_w);
//	checkCUDNN(cudnnSetTensor4dDescriptor(sftTensorDesc, CUDNN_TENSOR_NCHW, CUDNN_DATA_FLOAT, out_n, out_c, out_h, out_w));
//
//	//입력과 필터, 컨벌루션 패딩, 스트라이드가 주어졌을 때 가장 빠른 알고리즘 찾기
//	cudnnConvolutionFwdAlgo_t algo;
//	checkCUDNN(cudnnGetConvolutionForwardAlgorithm(cudnnHandle,
//		inTensorDesc,
//		filterDesc,
//		convDesc,
//		outTensorDesc,
//		CUDNN_CONVOLUTION_FWD_PREFER_FASTEST,
//		0,
//		&algo));
//
//	cout << "Fatest algorithm for conv0 = " << algo << endl;
//
//	cudnnConvolutionFwdAlgo_t algo2;
//	checkCUDNN(cudnnGetConvolutionForwardAlgorithm(cudnnHandle,
//		poolOutTensorDesc,
//		filterDesc2,
//		convDesc2,
//		sftTensorDesc,
//		CUDNN_CONVOLUTION_FWD_PREFER_FASTEST,
//		0,
//		&algo2));
//
//	cout << "Fatest algorithm for conv1 = " << algo2 << endl;
//
//	//위의 빠른 알고리즘을 사용할 경우 필요한 버퍼 데이터 크기 알아내기
//	size_t sizeinBytes = 0;
//	checkCUDNN(cudnnGetConvolutionForwardWorkspaceSize(cudnnHandle,
//		inTensorDesc,
//		filterDesc,
//		convDesc,
//		outTensorDesc,
//		algo,
//		&sizeinBytes));
//
//	cout << "workspace size (sizeinbytes):" << sizeinBytes << endl;
//	//계산 과정에서 버퍼 메모리가 필요한 경우 메모리 할당
//	if (sizeinBytes != 0) checkCUDA(cudaMalloc(&workSpace, sizeinBytes));
//
//	float alpha = 1.0f;
//	float beta = 0.0f;
//	//컨벌루션 시작
//	//"output = alpha * Op(input) + beta* output
//	//컨벌루션은 "output = 1 * Op(inpuit) + 0 * output"
//
//
//	checkCUDNN(cudnnConvolutionForward(cudnnHandle,
//		&alpha,
//		inTensorDesc,
//		inData_d,
//		filterDesc,
//		filterData_d,
//		convDesc,
//		algo,
//		workSpace,
//		sizeinBytes,
//		&beta,
//		outTensorDesc,
//		outData_d));
//
//	checkCUDA(cudaMemcpy(hostArray, outData_d, sizeof(float)* src_len, cudaMemcpyDeviceToHost));
//	print("conv out", hostArray, filter_num, in_height, in_width);
//
//	//add bias
//	beta = 1.0f;
//	checkCUDNN(cudnnAddTensor(cudnnHandle,
//		&alpha,
//		biasTensorDesc,
//		biasData_d,
//		&beta,
//		outTensorDesc,
//		outData_d));
//	checkCUDA(cudaMemcpy(hostArray, outData_d, sizeof(float)* src_len, cudaMemcpyDeviceToHost));
//	print("add bias out", hostArray, filter_num, in_height, in_width);
//
//	//activation_RELU
//	beta = 0.0f;
//	checkCUDNN(cudnnActivationForward(cudnnHandle,
//		actDesc,
//		&alpha,
//		outTensorDesc,
//		outData_d,
//		&beta,
//		outTensorDesc,
//		outData1_d));
//	//checkCUDA(cudaMemcpy(hostArray, outData1_d, sizeof(float)* src_len, cudaMemcpyDeviceToHost));
//	//print("RELU out", hostArray, filter_num, in_height, in_width);
//
//	//pooling
//	checkCUDNN(cudnnPoolingForward(cudnnHandle,
//		poolDesc,
//		&alpha,
//		outTensorDesc,
//		outData1_d,
//		&beta,
//		poolOutTensorDesc,
//		outData_d));
//	//checkCUDA(cudaMemcpy(hostArray, outData_d, sizeof(float)*pool_len, cudaMemcpyDeviceToHost));
//	//print("POOLING Out", hostArray, filter_num, pool_h, pool_w);
//
//	cublasSgemm(cublasHandle, CUBLAS_OP_N, CUBLAS_OP_N, 4, 4, out_channel, );
//
//	checkCUDA(cudaMemcpy(hostArray, outData1_d, sizeof(float)*out_channel, cudaMemcpyDeviceToHost));
//	print("FC Out:", hostArray, out_channel, 1, 1);
//
//	//softmax
//	checkCUDNN(cudnnSoftmaxForward(cudnnHandle,
//		CUDNN_SOFTMAX_ACCURATE,
//		CUDNN_SOFTMAX_MODE_CHANNEL,
//		&alpha,
//		sftTensorDesc,
//		outData1_d,
//		&beta,
//		sftTensorDesc,
//		outData_d));
//
//	checkCUDA(cudaMemcpy(hostArray, outData_d, sizeof(float)*out_channel, cudaMemcpyDeviceToHost));
//	print("softmax out", hostArray, out_channel, 1, 1);
//
//
//	Sleep(1000000);
//
//	//메모리 해제
//	checkCUDNN(cudnnDestroyTensorDescriptor(inTensorDesc));
//	checkCUDNN(cudnnDestroyTensorDescriptor(outTensorDesc));
//	checkCUDNN(cudnnDestroyFilterDescriptor(filterDesc));
//	checkCUDNN(cudnnDestroyFilterDescriptor(filterDesc2));
//	checkCUDNN(cudnnDestroyConvolutionDescriptor(convDesc));
//	checkCUDNN(cudnnDestroyConvolutionDescriptor(convDesc2));
//	checkCUDNN(cudnnDestroyTensorDescriptor(biasTensorDesc));
//	checkCUDNN(cudnnDestroyTensorDescriptor(poolOutTensorDesc));
//	checkCUDNN(cudnnDestroyTensorDescriptor(sftTensorDesc));
//	checkCUDNN(cudnnDestroyPoolingDescriptor(poolDesc));
//	checkCUDNN(cudnnDestroyActivationDescriptor(actDesc));
//	checkCUDNN(cudnnDestroy(cudnnHandle));
//
//	checkCUDA(cudaFree(inData_d));
//	checkCUDA(cudaFree(outData_d));
//	checkCUDA(cudaFree(outData1_d));
//	checkCUDA(cudaFree(filterData2_d));
//	checkCUDA(cudaFree(filterData_d));
//	checkCUDA(cudaFree(biasData_d));
//	cublasDestroy(cublasHandle);
//
//	checkCUDA(cudaThreadSynchronize());
//	return 0;
//
//}